#include "hip/hip_runtime.h"
#include <bitset>

#include <cuda/std/array>
#include <hiprand/hiprand_kernel.h>
#include <fmt/ranges.h>
#include <fmt/core.h>

#include "mma.cuh"
#include "util.cuh"

__host__ __forceinline__
void bitManip() {
    unsigned int x = 1U;
    std::cout << (x >> 0 & 1) << std::endl;
    std::cout << (x >> 31 & 1) << std::endl;
    x |= 1U << 31U;
    std::cout << std::bitset<32>(x) << std::endl;
    std::cout << (x >> 31 & 1) << std::endl;
}

template<unsigned int seed = 42, typename R>
requires(cuda::std::is_same_v<R, hiprandState> ||
    cuda::std::is_same_v<R, hiprandState_t> || cuda::std::is_same_v<R, hiprandStatePhilox4_32_10_t>)
__global__ void randK(R* __restrict__ states, void* __restrict__ rN) {
    auto* __restrict__ rN4 = CAST_TO(float4, rN);
    auto* __restrict__ cS = states + threadIdx.x;
    hiprand_init(seed, threadIdx.x, 0, cS);
    rN4[threadIdx.x] = hiprand_uniform4(cS);
}


int main() {
    constexpr auto N = 128;
    using R = hiprandStatePhilox4_32_10_t;
    R *dS;
    float* dR;
    using Element = float;
    cuda::std::array<Element, N * 4> hR{};
    CHECK_ERROR_EXIT(hipMallocAsync(&dS, N * sizeof(R), hipStreamPerThread));
    CHECK_ERROR_EXIT(hipMallocAsync(&dR, hR.size() * sizeof(Element), hipStreamPerThread));
    randK<<<1, N, 0, hipStreamPerThread>>>(dS, dR);
    CHECK_ERROR_EXIT(hipMemcpyAsync(hR.data(), dR, sizeof(Element) * hR.size(),
        hipMemcpyDeviceToHost, hipStreamPerThread));
    CHECK_ERROR_EXIT(hipPeekAtLastError());
    CHECK_ERROR_EXIT(hipStreamSynchronize(hipStreamPerThread));
    cuda::std::array<Element, 16> fHR{};
    std::memcpy(fHR.data(), hR.data(), sizeof(Element) * fHR.size());
    fmt::println("{}", fHR);
    //bitManip();
    //testCollective();
}
