#include "hip/hip_runtime.h"
#include <iostream>

#include <boost/random/uniform_int_distribution.hpp>
#include <boost/random/mersenne_twister.hpp>

#include <cooperative_groups/memcpy_async.h>
#include <cuda/cmath>
#include <cuda/std/array>
#include <cuda/std/chrono>
#include <cutlass/epilogue/thread/activation.h>
#include "cutlass/gemm/dispatch_policy.hpp"
#include "cutlass/gemm/collective/collective_mma.hpp"
#include <fmt/ranges.h>
#include <hip/hip_runtime.h>
#include <nccl.h>
#include <nvshmemx.h>
#include <nvshmem.h>
#include <host/nvshmemx_api.h> // Makes CLion happy
#include "processor/gemm.cuh"
#include "processor/tiling.cuh"

#define CAST_TO(T, p) static_cast<T*>(static_cast<void*>(p))
#define BYTE_CAST(p) static_cast<cuda::std::byte*>(static_cast<void*>(p))

#define NANO_TO_MICRO (cuda::std::nano::den / cuda::std::micro::den)
#if !defined(CHECK_ERROR_EXIT)
#  define CHECK_ERROR_EXIT(e)                                         \
do {                                                           \
    hipError_t code = (e);                                      \
    if (code != hipSuccess) {                                   \
    fprintf(stderr, "<%s:%d> %s:\n    %s: %s\n",               \
    __FILE__, __LINE__, #e,                            \
    hipGetErrorName(code), hipGetErrorString(code)); \
    fflush(stderr);                                            \
    exit(1);                                                   \
    }                                                            \
} while (0)
#endif

constexpr unsigned int len = 10000U;
template<unsigned int n>
__device__ __forceinline__
unsigned int blockManipulation(const cuda::std::array<bool, n>& isRemote,
    const unsigned int& idx) {
    unsigned int numPeers = 0U;
    cuda::std::array<unsigned int, n> peers{};
#pragma unroll
    for(unsigned int i = 0U; i < n; ++i) {
        const bool b = (idx > 0) * !isRemote[i] + isRemote[i] * (idx == 0);
        peers[numPeers] = !b * peers[numPeers] + i * b;
        numPeers += b;
    }
    return numPeers;
}

template<unsigned int n>
__device__ __forceinline__
unsigned int blockManipulationBranch(const cuda::std::array<bool, n>& isRemote,
    const unsigned int& idx) {
    unsigned int numPeers = 0U;
    cuda::std::array<unsigned int, n> peers{};
#pragma unroll
    for(unsigned int i = 0U; i < n; ++i) {
        if ((isRemote[i] && idx == 0) || (!isRemote[i] && idx > 0)) {
            peers[numPeers++] = i;
        }
    }
    return numPeers;
}

template<unsigned int n>
__global__ void benchBranch(const bool* in, __grid_constant__ const unsigned int idx) {
    cuda::std::array<bool, n> isRemote{};
    size_t start, end;
    double duration = 0.0;
#pragma unroll
    for (unsigned int i = 0; i < n; ++i) {
        isRemote[i] = in[i];
    }
    constexpr unsigned int runs = 4;
#pragma unroll
    for (unsigned int i = 0; i < runs; ++i) {
        asm volatile("mov.u64 %0, %%globaltimer;": "=l"(start)::);
        blockManipulation<len>(isRemote, idx);
        asm volatile("mov.u64 %0, %%globaltimer;": "=l"(end)::);
        duration += static_cast<double>(end - start) / static_cast<double>(runs);
    }
    printf("Branch less is %f, res is %u\n", duration, blockManipulation<len>(isRemote, idx));
    duration = 0.0;
#pragma unroll
    for (unsigned int i = 0; i < runs; ++i) {
        asm volatile("mov.u64 %0, %%globaltimer;": "=l"(start)::);
        blockManipulationBranch<len>(isRemote, idx);
        asm volatile("mov.u64 %0, %%globaltimer;": "=l"(end)::);
        duration += static_cast<double>(end - start) / static_cast<double>(runs);
    }
    printf("Branch is %f, res is %u\n", duration, blockManipulationBranch<len>(isRemote, idx));
}

__always_inline
void launchBenchBranch() {
    boost::random::mt19937 rng(cuda::std::chrono::high_resolution_clock::now()
        .time_since_epoch().count());
    const boost::random::uniform_int_distribution<> bits(0,1);
    std::array<bool, len> b{};
    for (unsigned int i = 0; i < len; ++i) {
        b[i] = bits(rng);
    }
    //fmt::println("{}", b);

    bool* bDevice;
    constexpr unsigned int idx = 1U;
    CHECK_ERROR_EXIT(hipMalloc(&bDevice, sizeof(bool)*len));
    CHECK_ERROR_EXIT(hipMemcpy(bDevice, b.data(), sizeof(bool)*len, hipMemcpyHostToDevice));
    benchBranch<len><<<1,1>>>(bDevice, idx);
    CHECK_ERROR_EXIT(hipPeekAtLastError());
    CHECK_ERROR_EXIT(hipDeviceSynchronize());
}

struct __align__(16) Args {
    double* sHeap;
    uint64_t* flags;
    double* result;
    unsigned int n;
    unsigned int rank;
    bool remotePresent;
    unsigned int processingRate;

    Args() = default;
    Args(double* _sHeap, uint64_t * _flags,
        double* _result, const unsigned int& _n,
        const unsigned int& _rank, const bool& _remotePresent, const unsigned int& _processingRate)
        : sHeap(_sHeap),
          flags(_flags),
          result(_result),
          n(_n),
          rank(_rank),
          remotePresent(_remotePresent),
          processingRate(_processingRate) {}
};

__constant__ Args b{};
void __global__ testArgs() {
    printf("Args has rank %u, results %f\n", b.rank, b.result[0]);
    b.sHeap[0] = 45.0;
    b.result[0] = 59.0;
    printf("Args has rank %u, results %f\n", b.rank, b.result[0]);
}
#define TO_MB(b) static_cast<double>(b) / (1024.0f*1024.0f)
void testArgsHost() {
    void* p;
    CHECK_ERROR_EXIT(hipMalloc(&p, sizeof(double)*4));
    CHECK_ERROR_EXIT(hipMemset(p, 0, sizeof(double)*4));
    const auto a = Args(static_cast<double*>(p),
        static_cast<uint64_t *>(p) + 1,
        static_cast<double*>(p) + 2,
        1, 0, true, 1);
    CHECK_ERROR_EXIT(hipMemcpyToSymbol(HIP_SYMBOL(b), &a, sizeof(Args)));
    testArgs<<<1,1>>>();
    CHECK_ERROR_EXIT(hipPeekAtLastError());
    CHECK_ERROR_EXIT(hipDeviceSynchronize());
    std::cout << TO_MB(1024*1024) << std::endl;
}

enum signal : unsigned short {
    NOOP,
    shouldProcess,
    processed,
};

#define STAGES 2U
#define CELLS 2U
template<unsigned int stage=0, typename T>
// Pointer arithmetic on void yields undefined behaviour
requires (stage < STAGES && !cuda::std::is_same_v<T, void>)
CUTE_DEVICE
T* advanceHeap(T* const& __restrict__ buffer, const unsigned int& slotSize,
    const unsigned int& peer) {
    return buffer + slotSize * ((STAGES * peer) + stage);
}

//cublasdx::sm_of<BLAS>::value
template<class GEMM, unsigned short rounds, bool skip=true>
requires (cublasdx::is_complete_blas_execution<GEMM>::value
&& cublasdx::is_supported<GEMM, cublasdx::sm_of<GEMM>::value>::value)
__global__ void overlapKernel(const typename GEMM::a_value_type* __restrict__ inputs,
    const typename GEMM::b_value_type* __restrict__ weights, cuda::std::byte* __restrict__ staging,
    uint64_t* __restrict__ flags, cuda::std::byte* sHeap, __grid_constant__ const unsigned int rank,
    __grid_constant__ const unsigned int world) {
    // The workflow operates as follows,
    // assuming each PE has a weight matrix and starts with an input matrix.
    // 1. At time i A2A to disseminate vector v_i
    // 2. GEMM on all received vectors
    // 3. A2A to reconstitute original vector v_i
    // 3. Process received vector
    // 4. Repeat
    assert(world == gridDim.x);
    assert(gridDim.y * gridDim.z == 1);
    static_assert(signal::processed == STAGES);
    static_assert(cublasdx::size_of<GEMM>::n == cublasdx::size_of<GEMM>::k);

    extern __shared__ __align__(16) char workspace[];
    __shared__ unsigned int bid;
    // Ensures a 32-bit single register is used
    const unsigned int tid = cooperative_groups::thread_block::thread_rank();
    constexpr auto sliceBytes = sizeof(GEMM::c_value_type) * GEMM::c_size;
    if (tid == 0) {
        // grid::block_rank() == peer rank
        bid = cooperative_groups::grid_group::block_rank();
        staging += sliceBytes * bid;
    }
    __threadfence_block();
    __syncthreads();

    // Make global memory tensor
    auto tAgA = cublasdx::make_tensor(inputs, GEMM::get_layout_gmem_a());
    auto tBgB = cublasdx::make_tensor(weights, GEMM::get_layout_gmem_b());
    auto tCgC = cublasdx::make_tensor(inputs, GEMM::get_layout_gmem_c());
    auto [sA, sB, sC] = GEMM::slice_shared_memory(workspace);

    // Make shared memory tensor
    auto tAsA = cublasdx::make_tensor(sA, GEMM::suggest_layout_smem_a());
    auto tBsB = cublasdx::make_tensor(sB, GEMM::suggest_layout_smem_b());
    auto tCsC = cublasdx::make_tensor(sC, GEMM::suggest_layout_smem_c());

    // Load data from global memory tensor to shared memory tensor
    // Note each block has identical copy of weights
    cublasdx::copy<GEMM, cublasdx::suggested_alignment_of<GEMM>::a_alignment>(tAgA, tAsA);
    cublasdx::copy<GEMM, cublasdx::suggested_alignment_of<GEMM>::b_alignment>(tBgB, tBsB);
    cublasdx::copy<GEMM, cublasdx::suggested_alignment_of<GEMM>::c_alignment>(tCgC, tCsC);
    cublasdx::copy_wait();
#if 0
    if (tid == 0) {
        print_tensor(tAsA);
        print_tensor(tBsB);
    }
#endif

    CUTE_UNROLL
    for (unsigned short i = 0; i < rounds; ++i) {
        // upper bound of number of messages per round
        memcpy_async(cooperative_groups::this_thread_block(), staging, BYTE_CAST(sC), sliceBytes);
        wait(cooperative_groups::this_thread_block());
        // Communicate vector to peer
        nvshmemx_putmem_signal_nbi_block(advanceHeap<0>(sHeap, sliceBytes, rank),
            staging, sliceBytes, flags + rank, shouldProcess, NVSHMEM_SIGNAL_SET, bid);
        if (!tid) {
            // Await data arrival
            nvshmem_signal_wait_until(flags + bid, NVSHMEM_CMP_EQ, shouldProcess);
        }
        __syncthreads();

        /// First stage
        // Copy received data to shared memory workspace
        memcpy_async(cooperative_groups::this_thread_block(), BYTE_CAST(sA),
            advanceHeap<0>(sHeap, sliceBytes, bid), sliceBytes);
        wait(cooperative_groups::this_thread_block());
        // Execute GEMM
        GEMM().execute(GEMM::a_value_type(1.0), tAsA, tBsB, GEMM::c_value_type(0.0), tCsC);
        __syncthreads();

#if 0
        if (tid == 0 and bid == 1 and rank == 0) {
            print_tensor(tAsA);
            print_tensor(tBsB);
            print_tensor(tCsC);
        }
#endif

        memcpy_async(cooperative_groups::this_thread_block(), staging, BYTE_CAST(sC), sliceBytes);
        wait(cooperative_groups::this_thread_block());

        // Eagerly communicate computed vector to peer
        nvshmemx_putmem_signal_nbi_block(advanceHeap<1>(sHeap, sliceBytes, rank),
            staging, sliceBytes, flags + world + rank, processed, NVSHMEM_SIGNAL_SET, bid);

        // Second Stage
        if (!tid) {
            // Await data arrival
            nvshmem_signal_wait_until(flags + world + bid, NVSHMEM_CMP_EQ, processed);
        }
        __syncthreads();
        memcpy_async(cooperative_groups::this_thread_block(), BYTE_CAST(sA),
            advanceHeap<1>(sHeap, sliceBytes, bid), sliceBytes);
        wait(cooperative_groups::this_thread_block());

        // Fused GEMM and ReLU
        GEMM().execute(GEMM::a_value_type(1.0), tAsA, tBsB, GEMM::c_value_type(0.0), tCsC,
            cublasdx::identity{}, cublasdx::identity{}, cublasdx::identity{},
            cutlass::epilogue::thread::ReLU<typename GEMM::c_value_type>{});
        __syncthreads();
#if 0
        if (tid == 0 and bid == 1 and rank == 0) {
            print_tensor(tAsA);
            print_tensor(tCsC);
        }
#endif
    }

    // Store final result in global memory, reusing staging
    memcpy_async(cooperative_groups::this_thread_block(), staging, BYTE_CAST(sC), sliceBytes);
}

void overlapPrototype() {
    auto playStream = hipStreamPerThread;
    // construct GEMM description
    constexpr auto M = 2U;
    constexpr auto N = 2U;
    constexpr auto K = 2U;
    using inputValueType = float;
    using weightValueType = float;
    using outValueType = float;
    // Do y=xA^T
    using GEMM = decltype(cublasdx::Size<M, N, K>()
                          + cublasdx::Precision<inputValueType>()
                          + cublasdx::Type<cublasdx::type::real>()
                          + cublasdx::Arrangement<cublasdx::row_major>()
                          + cublasdx::Function<cublasdx::function::MM>()
                          + cublasdx::SM<800>()
                          + cublasdx::Block());

    // blocks should be equal to n
    nvshmem_init();
    const auto nPes = nvshmem_n_pes();
    const auto rank = nvshmem_my_pe();
    CUTE_CHECK_ERROR(hipSetDevice(nvshmem_team_my_pe(NVSHMEMX_TEAM_NODE)));
    const auto abSize = (sizeof(GEMM::a_value_type) * GEMM::a_size * nPes)
    + (sizeof(GEMM::b_value_type) * GEMM::b_size);
    cuda::std::byte* ab;
    CUTE_CHECK_ERROR(hipMallocAsync(&ab, abSize, playStream));
    const auto heapBytes = (sizeof(GEMM::c_value_type) * GEMM::c_size * nPes * (STAGES + 1))
    + (sizeof(uint64_t) * nPes * STAGES);
    static_assert(sizeof(cuda::std::byte) == sizeof(unsigned char));
    auto* sHeap = static_cast<cuda::std::byte*>(nvshmem_calloc(heapBytes, sizeof(cuda::std::byte)));

    auto* data = malloc(abSize);
    int i = 0;
    static_assert(sizeof(inputValueType) == sizeof(weightValueType));
    static_assert(sizeof(inputValueType) == sizeof(outValueType));
    static_assert(sizeof(weightValueType) == sizeof(outValueType));
    auto bookend = GEMM::a_size * nPes;
    for (;i < bookend; ++i) {
        static_cast<inputValueType*>(data)[i] = static_cast<inputValueType>(rank + i);
    }
    bookend = i + GEMM::b_size;
    for (int j = 0; i < bookend; ++i, ++j) {
        static_cast<weightValueType*>(data)[i] = static_cast<weightValueType>(rank + j + 4);
    }

    CUTE_CHECK_ERROR(hipMemcpyAsync(ab, data, abSize, hipMemcpyHostToDevice, playStream));
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    CUTE_CHECK_ERROR(hipEventRecord(start, playStream));
    overlapKernel<GEMM, 1><<<nPes, GEMM::suggested_block_dim,
    GEMM::shared_memory_size, playStream>>>(
        CAST_TO(inputValueType, ab),
        CAST_TO(weightValueType, ab + (sizeof(GEMM::a_value_type) * GEMM::a_size * nPes)),
        sHeap,
        CAST_TO(uint64_t, sHeap + (GEMM::c_size * nPes * sizeof(GEMM::c_value_type))),
        sHeap + (GEMM::c_size * nPes * sizeof(GEMM::c_value_type)) + (sizeof(uint64_t) * nPes * STAGES),
        nvshmem_my_pe(), nPes);
    CUTE_CHECK_ERROR(hipEventRecord(stop, playStream));
    CUTE_CHECK_ERROR(hipPeekAtLastError());
    CUTE_CHECK_ERROR(hipStreamSynchronize(playStream));
    float duration = 0.0f;
    CUTE_CHECK_ERROR(hipEventElapsedTime(&duration, start, stop));
    fmt::println("Elapsed time {}", duration);
    // Copy matrix C
    CUTE_CHECK_ERROR(hipMemcpyAsync(data, sHeap, sizeof(GEMM::c_value_type) * GEMM::c_size * nPes,
        hipMemcpyDeviceToHost, playStream));
    CUTE_CHECK_ERROR(hipPeekAtLastError());
    CUTE_CHECK_ERROR(hipStreamSynchronize(playStream));
    nvshmem_free(sHeap);
    nvshmem_finalize();
    CUTE_CHECK_ERROR(hipEventDestroy(start));
    CUTE_CHECK_ERROR(hipEventDestroy(stop));
    CUTE_CHECK_ERROR(hipFreeAsync(ab, playStream));
    CUTE_CHECK_ERROR(hipPeekAtLastError());
    CUTE_CHECK_ERROR(hipStreamSynchronize(playStream));
    // print result
    if (rank == 0) {
        print_tensor(make_tensor(static_cast<GEMM::c_value_type*>(data), cute::make_shape(M*nPes, N)));
    }
    free(data);
}

void testGEMM() {
    introduction_example<800>();
}

void testConfig() {
    constexpr auto M = 2U;
    constexpr auto N = 2U;
    constexpr auto K = 2U;
    using inputValueType = cublasdx::tfloat32_t;
    using weightValueType = cublasdx::tfloat32_t;
    using outValueType = float;
    // Do y=xA^T
    using GEMM = decltype(cublasdx::Size<M, N, K>()
                          + cublasdx::Precision<inputValueType, weightValueType, outValueType>()
                          + cublasdx::Type<cublasdx::type::real>()
                          + cublasdx::Arrangement<cublasdx::row_major>()
                          + cublasdx::Function<cublasdx::function::MM>()
                          + cublasdx::SM<800>()
                          + cublasdx::Block());
    constexpr bool isALayoutLeft = cublasdx::arrangement_of<GEMM>::a == cublasdx::arrangement::col_major;
    constexpr bool isBLayoutLeft = cublasdx::arrangement_of<GEMM>::b == cublasdx::arrangement::col_major;
    constexpr bool isCLayoutLeft = cublasdx::arrangement_of<GEMM>::c == cublasdx::arrangement::col_major;
    using optimalConfig = cublasdx::detail::layout_database::optimal_config<128, 800,
    inputValueType, isALayoutLeft, cublasdx::alignment_of<GEMM>::a,
    weightValueType, isBLayoutLeft, cublasdx::alignment_of<GEMM>::b,
    outValueType, isCLayoutLeft, cublasdx::alignment_of<GEMM>::c,
    M, N, K>;
}

template<class BlockMM, class ProblemShape>
requires (cublasdx::is_complete_blas<BlockMM>::value
&& cublasdx::is_supported<BlockMM, cublasdx::sm_of<BlockMM>::value>::value)
__global__ void testCollectiveMMA(ProblemShape shapeMNK,
    const typename BlockMM::a_value_type* __restrict__ inputs,
    const typename BlockMM::b_value_type* __restrict__ weights,
    typename BlockMM::c_value_type* __restrict__ result) {
    constexpr bool isALayoutLeft = cublasdx::arrangement_of<BlockMM>::a == cublasdx::arrangement::col_major;
    constexpr bool isBLayoutLeft = cublasdx::arrangement_of<BlockMM>::b == cublasdx::arrangement::col_major;
    constexpr bool isCLayoutLeft = cublasdx::arrangement_of<BlockMM>::c == cublasdx::arrangement::col_major;
    using optimalConfig = cublasdx::detail::layout_database::optimal_config<128, cublasdx::sm_of<BlockMM>::value,
    typename BlockMM::a_value_type, isALayoutLeft, cublasdx::alignment_of<BlockMM>::a,
    typename BlockMM::b_value_type, isBLayoutLeft, cublasdx::alignment_of<BlockMM>::b,
    typename BlockMM::c_value_type, isCLayoutLeft, cublasdx::alignment_of<BlockMM>::c,
    cublasdx::size_of<BlockMM>::m, cublasdx::size_of<BlockMM>::n, cublasdx::size_of<BlockMM>::k>;

    using Parameters = GEMMParameters<BlockMM>;
    constexpr auto bM = cublasdx::size_of<BlockMM>::m;
    constexpr auto bN = cublasdx::size_of<BlockMM>::n;
    constexpr auto bK = cublasdx::size_of<BlockMM>::k;
    using blockTiler = cute::Shape<cute::Int<bM>, cute::Int<bN>, cute::Int<bK>>;

    using CollectiveMainloop = typename cutlass::gemm::collective::CollectiveMma<
        cutlass::gemm::MainloopSm80CpAsyncUnpredicated<cute::_1::value>,
        blockTiler,
        typename BlockMM::a_value_type,
        cute::Underscore,
        typename BlockMM::b_value_type,
        cute::Underscore,
        typename Parameters::config::TiledMma,
        typename Parameters::gCopyA,
        typename Parameters::config::a_layout,
        typename Parameters::config::a_copy_op,
        cute::identity,
        typename Parameters::gCopyB,
        typename Parameters::config::b_layout,
        typename Parameters::config::b_copy_op,
        cute::identity
    >;

    typename Parameters::config::TiledMma tiledMMA;
    using TilerOut = cute::Shape<cute::Int<bM>, cute::Int<bN>>;
    auto accum = cute::partition_fragment_C(tiledMMA, TilerOut{});

    // Represent the full tensors
    auto mA = cute::make_tensor(cute::make_gmem_ptr(inputs), cute::select<0,2>(shapeMNK), Parameters::strideA{}); // (M,K)
    auto mB = cute::make_tensor(cute::make_gmem_ptr(weights), cute::select<1,2>(shapeMNK), Parameters::strideB{}); // (N,K)
    auto mC = cute::make_tensor(cute::make_gmem_ptr(result), cute::select<0,1>(shapeMNK), Parameters::strideC{}); // (M,N)


    // Get the appropriate blocks for this thread block
    auto cta_coord = make_coord(blockIdx.x, blockIdx.y, cute::_);              // (m,n,k)
    auto gA = local_tile(mA, blockTiler{}, cta_coord, cute::Step<cute::_1, cute::X,cute::_1>{});  // (BLK_M,BLK_K,k)
    auto gB = local_tile(mB, blockTiler{}, cta_coord, cute::Step< cute::X,cute::_1,cute::_1>{});  // (BLK_N,BLK_K,k)
    auto gC = local_tile(mC, blockTiler{}, cta_coord, cute::Step<cute::_1,cute::_1, cute::X>{});  // (BLK_M,BLK_N)

    auto k_tile_iter = cute::make_coord_iterator(size<2>(gA));
    int k_tile_count = size<2>(gA);


}

template<unsigned int Arch>
__global__ void testArch() {
    printf("%u", 5);
}

void testAlloc() {
    nvshmem_init();
    CUTE_CHECK_ERROR(hipSetDevice(nvshmem_team_my_pe(NVSHMEMX_TEAM_NODE)));
    auto* p = nvshmem_calloc(4,1);
    auto* pA = nvshmem_malloc(4);
    auto* pAlign = nvshmem_align(16, 4);
    std::cout << ((uintptr_t)p % 16 == 0) << std::endl;
    std::cout << ((uintptr_t)pA % 16 == 0) << std::endl;
    std::cout << ((uintptr_t)pAlign % 16 == 0) << std::endl;
    std::cout << ((uintptr_t)p) << std::endl;
    std::cout << ((uintptr_t)pA) << std::endl;
    std::cout << ((uintptr_t)pAlign) << std::endl;
    nvshmem_free(p);
    nvshmem_free(pA);
    nvshmem_free(pAlign);
    nvshmem_finalize();
}

int main() {
    auto b = cute::Shape<cute::_128, cute::_8>{};
    auto s = cute::Shape<cute::_32,cute::_8>{};
    constexpr auto v = cute::_32::value;
    //overlapPrototype();
    return 0;
}